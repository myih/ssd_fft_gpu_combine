#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include "include/ssd_fft_gpu.h"

#define IMUL(a, b) __mul24(a, b)

#ifdef US_SIGNS
	//dead rows starts at the first row, no need to add +1 in iDataIndx calculation in fixDeadPixels kernel
	__device__ __constant__ int iAddOneRow = 0;
#else
	//in EU videos, dead rows start at second row, need to add +1 in iDataIndx calculation in fixDeadPixels kernel
	__device__ __constant__ int iAddOneRow = 1;
#endif

//constant var should be in file scope that is why I got rid of ssd_fft_kernel.h 
//and instead included ssd_fft_kernel.cu in the main.cu (had to deleted customBuild line from
//proj file)
__device__ __constant__ unsigned char d_acLUT[256];

//convert char to float and adjust contrast by doing gamma correction.
__global__ void convertChar4ToFloatDoConGam(uchar4* gd_ac4Scn, float4* d_afScn, int dataN, bool bConGam)
{
	int iIndx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	uchar4 c4DataIn;
	float4 f4DataOut;
	if (iIndx < dataN)
	{
	c4DataIn = gd_ac4Scn[iIndx];
	if (bConGam)
		{
		//doing ConGam takes 0.1 ms more
		f4DataOut.x = (float) d_acLUT[int(c4DataIn.x)];
		f4DataOut.y = (float) d_acLUT[int(c4DataIn.y)];
		f4DataOut.z = (float) d_acLUT[int(c4DataIn.z)];
		f4DataOut.w = (float) d_acLUT[int(c4DataIn.w)];
		}
	else
		{
		f4DataOut.x = (float) c4DataIn.x;
		f4DataOut.y = (float) c4DataIn.y;
		f4DataOut.z = (float) c4DataIn.z;
		f4DataOut.w = (float) c4DataIn.w;
		}
	d_afScn[iIndx] = f4DataOut;
	}
}

//fix dead pixel with averaging 8 immediate neighbors. 
__global__ void fixDeadPixels(hipfftReal* d_afScn, int iScnSz, int iScnW, int iScnH)
{
    __shared__ hipfftReal afTopRow[(BLOCKDIMX+(HALFWARP+1))];
    __shared__ hipfftReal afMidRow[(BLOCKDIMX+(HALFWARP+1))];
    __shared__ hipfftReal afBotRow[(BLOCKDIMX+(HALFWARP+1))];
	int iDeadRowDataIndx = IMUL(blockIdx.x, BLOCKDIMX) + (threadIdx.x-HALFWARP);
	int iDataIndx = iDeadRowDataIndx + IMUL((iDeadRowDataIndx/iScnW)+iAddOneRow,iScnW);
	
	afTopRow[threadIdx.x] = 0;
	afMidRow[threadIdx.x] = 0;
	afBotRow[threadIdx.x] = 0;

	if (iDataIndx >= 0 && iDataIndx < iScnSz && threadIdx.x >= HALFWARP-1 && threadIdx.x <= (blockDim.x-1))
	{
		int iRow = iDataIndx/iScnW;
		//read top row
		if (iRow > 0)
			afTopRow[threadIdx.x] = d_afScn[iDataIndx - iScnW];
		//read middle row
		afMidRow[threadIdx.x] = d_afScn[iDataIndx];
		//read bottom row
		if (iRow < iScnH-1)
			afBotRow[threadIdx.x] = d_afScn[iDataIndx + iScnW];
	}

	__syncthreads();
	
	if (iDataIndx >= 0 && iDataIndx < iScnSz && threadIdx.x > HALFWARP-1 && threadIdx.x < (blockDim.x-1))
	{
		hipfftReal fSum = 0;
		int iLeftIndx, iRightIndx;
		int iCol = iDataIndx%iScnW;
		if (iCol%2 == 0)
		{
			fSum = fSum + afTopRow[threadIdx.x] + afBotRow[threadIdx.x];
			int iNumNeigh = 2;
			if (iCol > 0)
			{
				iLeftIndx = threadIdx.x - 1 ;
				fSum = fSum + afTopRow[iLeftIndx] + afMidRow[iLeftIndx] + afBotRow[iLeftIndx];
				iNumNeigh = iNumNeigh + 3;
				//fSum = fSum + afMidRow[iLeftIndx];
				//iNumNeigh = iNumNeigh + 1;
			}
			if (iCol < iScnW-1)
			{
				iRightIndx = threadIdx.x + 1 ;
				fSum = fSum + afTopRow[iRightIndx] + afMidRow[iRightIndx] + afBotRow[iRightIndx];
				iNumNeigh = iNumNeigh + 3;
				//fSum = fSum + afMidRow[iRightIndx];
				//iNumNeigh = iNumNeigh + 1;
			}
		d_afScn[iDataIndx] = fSum / (float)iNumNeigh;
		}
	}
}

//take kth law of the data
__global__ void kthLaw(hipfftComplex* d_afPadScn, int dataN)
{
	int iIndx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if (iIndx < dataN)
	{
	//afVals(:) = (abs(afVals(:)).^k) .* (cos(angle(afVals(:))) + sin(angle(afVals(:)))*i);
	hipfftComplex cDat = d_afPadScn[iIndx];
	float fNewAbsDat = powf(sqrtf(powf(cDat.x,2)+ powf(cDat.y,2)),FK);
	float fAngDat = atan2f(cDat.y, cDat.x);
	cDat.x = fNewAbsDat*cosf(fAngDat);
	cDat.y = fNewAbsDat*sinf(fAngDat);
	d_afPadScn[iIndx] = cDat;
	}
}

__global__ void pointWiseMul(hipfftComplex* d_afCorr, hipfftComplex* d_afPadScn, hipfftComplex* d_afPadTpl,int dataN, float fScale)
{
	int iIndx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if (iIndx < dataN)
	{
	hipfftComplex cDat = d_afPadScn[iIndx];
	hipfftComplex cKer = d_afPadTpl[iIndx];
	//take the conjugate of the kernel
	cKer.y = -cKer.y; 
	hipfftComplex cMul = {cDat.x* cKer.x - cDat.y * cKer.y, cDat.y * cKer.x + cDat.x * cKer.y};
	//const float     q = 1.0f / (float)dataN;
    //cMul.x = q * cMul.x;
    //cMul.y = q * cMul.y;

    cMul.x = fScale * cMul.x;
    cMul.y = fScale * cMul.y;
	d_afCorr[iIndx] = cMul;
	}
}


// Complex scale
__global__ void ComplexScale(hipfftComplex* a, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
	{
    a[i].x = scale * a[i].x;
    a[i].y = scale * a[i].y;
	}
} 



/* Compute max with reduction (return key-value).
 */

__global__ void max_k(hipfftReal* afData, int* aiDataIdxs, int iSizeOfData, hipfftReal* afBlockMaxs, int* aiBlockMaxIdxs)
{

	// Block index
    int iBx = blockIdx.x;

    // Thread index
    int iTx = threadIdx.x;

	//Block dim
	int iBDimX = blockDim.x; //BLOCKDIMX_MAX
	__shared__ hipfftReal afSubMax[BLOCKDIMX_MAX];
	__shared__ int aiSubMaxIdx[BLOCKDIMX_MAX];

	int iIndx = iBx*(EACHTHREADREADS*iBDimX) + iTx;
	int iIdx;

	//each thread will read EACHTHREADREADS pixels and add them up
	afSubMax[iTx] = 0;
	aiSubMaxIdx[iTx] = -1;
	for (int i = 0; i < EACHTHREADREADS; i++)
		{
			iIdx = iIndx+(i*iBDimX); 
			if ( iIdx < iSizeOfData) 
			{
			afSubMax[iTx] = fmaxf(afSubMax[iTx], afData[iIdx]);
			if (afSubMax[iTx] == afData[iIdx])
				{
				if (aiDataIdxs == NULL)
					aiSubMaxIdx[iTx] = iIdx;
				else
					aiSubMaxIdx[iTx] = aiDataIdxs[iIdx];
				}
			}
		}
	__syncthreads();

	//this for loop does the reduce max!
	for (unsigned int d = iBDimX >> 1; d > 0; d >>= 1) 
    {
        if (iTx < d)
        {
			afSubMax[iTx] = fmaxf(afSubMax[iTx], afSubMax[iTx + d]);
			if (afSubMax[iTx] == afSubMax[iTx + d])
				aiSubMaxIdx[iTx] = aiSubMaxIdx[iTx + d];
        }
		__syncthreads();
    }

	if (iTx == 0) 
	{
		afBlockMaxs[iBx] = afSubMax[0];
		aiBlockMaxIdxs[iBx] = aiSubMaxIdx[0];
	}
}

