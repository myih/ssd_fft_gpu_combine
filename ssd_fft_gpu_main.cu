#include "hip/hip_runtime.h"

//#define US_SIGNS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
//#include <cutil.h>
#include "ssd_fft_gpu_kernel.cu"
//#define BUILD_DLL
#include <GL/glew.h>
#include <GL/glut.h>
//#include "include/ssd_fft_gpu_dll.h"
#include <ssd_fft_gpu_common.h>
#include "include/ssd_fft_gpu.h"
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>
#include <helper_timer.h>

#define CUTFalse false
#define CUTTrue true
#define CUTBoolean bool
#define CUDA_SAFE_CALL checkCudaErrors
#define CUFFT_SAFE_CALL checkCudaErrors
#define CUT_SAFE_CALL checkCudaErrors
#define CUT_CHECK_ERROR getLastCudaError
#define cutComparefe sdkCompareL2fe
#define cutCreateTimer sdkCreateTimer


extern "C"
int CLAHE(unsigned char* pImage, unsigned int uiXRes, unsigned int uiYRes, unsigned char Min,
	unsigned char Max, unsigned int uiNrX, unsigned int uiNrY,
	unsigned int uiNrBins, float fCliplimit);

#define gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx)  gd_afCompFlt + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) +  ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
#define d_pafWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx) d_pafWholeTplFFT + ((giScnHPad * giScnWPad * giNumIPInFirst * giNumSz) * iFltAbsIndx) + ((giScnHPad * giScnWPad * giNumIPInFirst) * iSzIndx) + ((giScnHPad * giScnWPad) * iIPAbsIndx)
#define gd_afWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx) gd_afWholeTplFFT + ((giScnHPad * giScnWPad * giNumIPInFirst * giNumSz) * iFltAbsIndx) + ((giScnHPad * giScnWPad * giNumIPInFirst) * iSzIndx) + ((giScnHPad * giScnWPad) * iIPAbsIndx)
#define d_pafPartTplFFT(iIPIndx, iSzIndx, iFltIndx) d_pafPartTplFFT + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) + ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
#define gd_afPartTplFFT(iIPIndx, iSzIndx, iFltIndx) gd_afPartTplFFT + ((giTplH * giTplW * giNumIPRot * giNumSz) * iFltIndx) + ((giTplH * giTplW * giNumIPRot) * iSzIndx) + ((giTplH * giTplW) * iIPIndx)
////////////////////////////////////////////////////////////////////////////////
// Global vars
////////////////////////////////////////////////////////////////////////////////
//trashold for the PSR (might be different for day and night)
//const float gfPSRTrashold = 8.0f;
const float gfPSRTrashold = 7.5f;
//params related to Majority Voting
//keep track of PSRs for giTrackingLen frames
const float giTrackingLen = 10;
float giFrameNo = 0;
int giNumFramesInAcc = 0; //number of frames that contribute to AccPSR
						  //max acc psr should be greater than gfAccPSRTrashold so that we can conclude that speed sign is recognized
float gfAccPSRTrashold = 0;
//factor which determines additional confidence due to IP (if IP is equal to prevIP increase conf). 
//makes sense when different IP Rots are defined.
const float gfAddConfIPFac = 0.25;
//factor which determines additional confidence due to Sz (if Sz is larger to prevSz increase conf). 
const float gfAddConfEqSzFac = 0.5;
const float gfAddConfGrSzFac = 1.25;

typedef struct AccRes_struct
{
	float fAccConf;
	int iPrevIP;
	int iPrevSz;
}AccRes_struct_t;

AccRes_struct_t* gastAccRes;


//scene dimension is constant 
/*#ifdef US_SIGNS
const int giScnBegY = 0;
const int giScnW = 640;
const int giScnH = 384;  //actually it is 390, but I make it 384 so that it is divisible by 8 and CLAHE works properly
const int giOrigScnSz = 640 * 390;
char acMeasure[5] = "mph";
#else
const int giScnBegY = 48; //48,0 y coordinate where the window begins (zero based index)
const int giScnW = 640; //window has tha same width as the scn.
const int giScnH = 240; //240,480 window height
const int giOrigScnSz = 640 * 480;
char acMeasure[5] = "km/h";
#endif*/

//for PSR calculation define sidelobe
//area = frame+mask
const int giAreaH = 20;
const int giMaskH = 4;

const int	giScnSz = giScnW * giScnH;
const int	giScnSzPad = giScnWPad * giScnHPad;
const int	giScnMemSzReal = giScnSz * sizeof(hipfftReal);
const int	giScnMemSzRealPad = giScnSzPad * sizeof(hipfftReal);//1024*1024
const int   giScnMemSzCmplx = giScnSz * sizeof(hipfftComplex);
const int   giScnMemSzCmplxPad = giScnSzPad * sizeof(hipfftComplex);//1024*1024
const int   giScnMemSzUChar = giScnSz * sizeof(unsigned char);
const int	giAreaMemSzReal = giAreaH * giAreaH * sizeof(hipfftReal);
const int	giScnOffset = giScnBegY * giScnW;
const int   giOrigScnMemSzUChar = giOrigScnSz * sizeof(unsigned char);

//directory where scene and templates are
char g_sPathBegin[50] = "cpuResults/";
char g_sPath[100];
//directory where stats files will be stored
char g_sStatsPathBegin[50] = "../stats/ssd_gpu_stats/fft_results/";
char g_sStatsPath[100];
FILE* g_fStatsFile;
//directory where scnbin files will be stored
#ifdef US_SIGNS
char g_sScnBinPathBegin[50] = "../convert_pgm_to_RawVideo/raw/";
#else
char g_sScnBinPathBegin[50] = "../copied15May17/EU_raw(savedRealisFilesAsBin)/";
#endif
char g_sScnBinPath[100];
FILE* g_fScnBin;

#ifdef REALTIME
unsigned long g_ulPrevTimeStamp = 0;
const int g_iRuntime = 124; //update this if you make performance improvements
const float	gfAccPSRTrasholdSpecialReal = 11.0f;
#endif

#ifdef STATS
unsigned long g_ulFirstTimeStamp = 0;
unsigned long g_ulLastTimeStamp = 0;
float g_fAllVideoTime = 0;
int g_iNumVideos = 0;
int gi16fps = 0;
int gi8fps = 0;
int gi5fps = 0;
int gi4fps = 0;
int gi0fps = 0; //infinity fps (time diff is 0ms)
#endif

				//unsigned int guiParTim;
StopWatchInterface *guiParTim;
//unsigned int guiKerTim;
StopWatchInterface *guiKerTim;
double g_dRunsOnGPUTotalTime;
double g_dTotalKerTime;
double g_dClaheTime;

int giTplH, giTplW, giTplSz, giTplWMemSz, giTplMemSzReal, giTplMemSzCmplx;
int giNumIPRot, giNumSz, giNumOrigFlt, giNumSngCompFlt;

typedef struct CompFlt_struct
{
	float* h_afData;
	int iH;
	int iW;
	int iNumIPRot;
	int iNumSz;
	int iNumOrigFlt;
	int iNumMulCompFlt;
	int iDataSz;
	int iDataMemSz;
	int* aiIPAngs;
	int* aiTplCols;
	int* aiTpl_no;
}CompFlt_struct_t;

CompFlt_struct_t gstCompFlt;

int giPartMaxGDx, giWholeMaxGDx;
hipfftReal
*gd_pfMax,
*gd_afBlockMaxs;
int
*gd_piMaxIdx,
*gd_aiBlockMaxIdxs;
////////////////////////////////////////////////////////////////////////////////
// Following variables have been made global, so that we can divide the main function
// to init, fingBestTpl, and exit
////////////////////////////////////////////////////////////////////////////////


//typedef float hipfftReal;
hipfftReal
*gd_afScnPartIn,
*gh_afArea,
*gd_afCompFlt,
*gd_afPadTplIn,
*gd_afPadScnIn,
*gd_afPadScnInPad,
*gd_afCorr;

//typedef float hipfftComplex[2];
hipfftComplex
*gd_afScnPartOut,
*gd_afPadTplOut,
*gd_afPadScnOut,
*gd_afPadScnOutPad,
*gd_afWholeTplFFT,
*gd_afPartTplFFT,
*gd_afMul;

unsigned char
*gh_acScn;

uchar4
*gd_ac4Scn;

hipfftHandle
ghFFTplanWholeFwd,
ghFFTplanWholeInv,
ghFFTplanPartFwd,
ghFFTplanPartInv;

dim3 gdThreadsConv(1, 1, 1);
dim3 gdBlocksConv(1, 1);
dim3 gdThreadsDead(1, 1, 1);
dim3 gdBlocksDead(1, 1);
dim3 gdThreadsWhole(1, 1, 1);
dim3 gdBlocksWhole(1, 1);
dim3 gdThreadsPart(1, 1, 1);
dim3 gdBlocksPart(1, 1);

int
giBegIdxIPInFirst,
giEndIdxIPInFirst,
giNumIPInFirst,
giBegIdxIPInSecond,
giEndIdxIPInSecond;

//adjust contrast and do gamma correction 
bool gbConGam = 0;
//fix the dead pixels in the given scene if we are processing a video 
bool gbFixDead = 1;

//params related to ConGam
#define LUTSIZE 256
float gfLUT[LUTSIZE];
unsigned char gacLUT[LUTSIZE];
float gfLIn = 0.2f;//0.4f;//0.2f;
float gfHIn = 0.8f;//0.6f;//0.8f;
float gfLOut = 0.0f;
float gfHOut = 1.0f;
float gfG = 2.5f;//0.5f;//2.5f;

				 //pass the found Speed Limit Number to the callee (GUI)
int giSLCurFrm = -1; //SL found in the current frame (-1 means no SL)
int giSLResult = -1; //SL found as a result of temporal integration (-1 means no SL)
int giShowClaheGUI = 0; //allow ssd_fft_GUI to turn on/off CLAHE showing (to capture the CLAHE effect in DAGM video) if -1 show, if 0 do not.
char gacClipName[11];
////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int iX, int iY) {
	return (iX % iY != 0) ? (iX / iY + 1) : (iX / iY);
}

//Align a to nearest higher multiple of b
int iAlignUp(int iX, int iY) {
	return (iX % iY != 0) ? (iX - iX % iY + iY) : iX;
}

//convert 1D Index to 2D Coordinates
void Indx2Coord(int iImgW, int iIndx, int* iRow, int* iCol)
{
	//cuda is row major and zero-based
	*iCol = iIndx%iImgW;
	//regular division of integer returns floor
	*iRow = iIndx / iImgW;
}

//convert 2D Coordinates to 2D Index
void Coord2Indx(int iImgW, int iRow, int iCol, int* iIndx)
{
	//cuda is row major and zero-based
	*iIndx = (iImgW*iRow) + iCol;
}

//assign values to rectangle specified by coord
void assignVal(int iImgW, float* afImg, int4 aiCoord, float fVal)
{
	int iIndx;
	for (int iRow = aiCoord.x; iRow <= aiCoord.y; iRow++)
	{
		for (int iCol = aiCoord.z; iCol <= aiCoord.w; iCol++)
		{
			Coord2Indx(iImgW, iRow, iCol, &iIndx);
			afImg[iIndx] = fVal;
		}
	}
}

//sum elements
float sum(float* afImg, int iSz)
{
	float fTotal = 0;
	for (int i = 0; i<iSz; i++)
	{
		fTotal += afImg[i];
	}
	return fTotal;
}

//get surrounding coordinates of the areas centered around a point 
int4 getSurrCoord(int iRow, int iCol, int iSurrH, int iNumCols, int iNumRows)
{
	//TODO: maybe I should shift area if it is at border, to produce lower PSR?
	int iHalfSurrH = iSurrH / 2;
	int iSurrRowBeg = iRow - iHalfSurrH + 1;
	if (iSurrRowBeg < 0) iSurrRowBeg = 0;
	int iSurrRowEnd = iRow + iHalfSurrH;
	if (iSurrRowEnd >= iNumRows) iSurrRowEnd = iNumRows - 1;
	int iSurrColBeg = iCol - iHalfSurrH + 1;
	if (iSurrColBeg < 0) iSurrColBeg = 0;
	int iSurrColEnd = iCol + iHalfSurrH;
	if (iSurrColEnd >= iNumCols) iSurrColEnd = iNumCols - 1;
	int4 aiAreaCoord = { iSurrRowBeg, iSurrRowEnd, iSurrColBeg, iSurrColEnd };

	return aiAreaCoord;
}

//make FFT size power of two 
int getPOTSz(int iSz) {
	//Highest non-zero bit position of iSz
	int iHiBit;
	//Neares lower and higher powers of two numbers for iSz
	unsigned int uiLowPOT, uiHiPOT;

	//Find highest non-zero bit (1U is unsigned one)
	for (iHiBit = 31; iHiBit >= 0; iHiBit--)
		if (iSz & (1U << iHiBit)) break;

	//No need to align, if already power of two
	uiLowPOT = 1U << iHiBit;
	if (uiLowPOT == iSz) return iSz;

	//Align to a nearest higher power of two, if the size is small enough,
	//else align only to a nearest higher multiple of 512,
	//in order to save computation and memory bandwidth
	uiHiPOT = 1U << (iHiBit + 1);
	if (uiHiPOT <= 1024)
		return uiHiPOT;
	else
		return iAlignUp(iSz, 512);
}


//Get the full path name
char* getFullPathOfFile(char* pcFileName)
{
	strcpy(g_sPath, g_sPathBegin);
	strcat(g_sPath, pcFileName);
	return g_sPath;
}

//compare GPU results to CPU results
void cmpCPU(void* afVals, char* pcFileName, bool bComplex, int iSz, bool bHost, float fEpsilon)
{
	int iMemSzReal = iSz * sizeof(hipfftReal);
	FILE* fRef = fopen(getFullPathOfFile(pcFileName), "rb");

	if (bComplex)
	{
		//file has both real and imaginary values
		int iMemSzCmplx = iSz * sizeof(hipfftComplex);
		hipfftComplex* h_afCmplx;
		if (!bHost)
		{
			h_afCmplx = (hipfftComplex *)malloc(iMemSzCmplx);
			// copy result from device to host
			CUDA_SAFE_CALL(hipMemcpy(h_afCmplx, (hipfftComplex*)afVals, iMemSzCmplx, hipMemcpyDeviceToHost));
		}
		else
		{
			h_afCmplx = (hipfftComplex *)afVals;
		}
		//extract real and imaginary parts
		float* h_afReal = (float*)malloc(iMemSzReal);
		float* h_afImag = (float*)malloc(iMemSzReal);
		for (int iI = 0; iI < iSz; iI++)
		{
			h_afReal[iI] = h_afCmplx[iI].x;
			h_afImag[iI] = h_afCmplx[iI].y;
		}

		// allocate mem to hold CPU results 
		float* afRealRef = (float*)malloc(iMemSzReal);
		float* afImagRef = (float*)malloc(iMemSzReal);
		fread(afRealRef, sizeof(float), iSz, fRef);
		fread(afImagRef, sizeof(float), iSz, fRef);

		CUTBoolean cutbResReal = cutComparefe(afRealRef, h_afReal, iSz, fEpsilon);
		CUTBoolean cutbResImag = cutComparefe(afImagRef, h_afImag, iSz, fEpsilon);
		printf("Checking %s result: %s\n", pcFileName, (1 == (cutbResReal && cutbResImag)) ? "PASSED" : "FAILED");
		//generate text file
#ifdef GENTXTOUTPUT
		FILE* fTxt = fopen(strcat(getFullPathOfFile(pcFileName), ".txt"), "w");
		for (int i = 0; i < iSz; i++)
		{
			fprintf(fTxt, "i = %d, My real: %f Ref real: % f - My imag: %f Ref Imag: %f\n", i, h_afReal[i], afRealRef[i], h_afImag[i], afImagRef[i]);
		}
		fclose(fTxt);
#endif
		//clean up memory
		if (!bHost) { free(h_afCmplx); }
		free(h_afReal);
		free(h_afImag);

		free(afRealRef);
		free(afImagRef);
	}
	else
	{
		//file has only real values
		int iMemSzReal = iSz * sizeof(hipfftReal);
		hipfftReal* h_afReal;
		if (!bHost)
		{
			h_afReal = (hipfftReal *)malloc(iMemSzReal);
			// copy result from device to host
			CUDA_SAFE_CALL(hipMemcpy(h_afReal, (hipfftReal*)afVals, iMemSzReal, hipMemcpyDeviceToHost));
		}
		else
		{
			h_afReal = (hipfftReal *)afVals;
		}

		// allocate mem to hold CPU results 
		float* afRealRef = (float*)malloc(iMemSzReal);
		fread(afRealRef, sizeof(float), iSz, fRef);

		CUTBoolean cutbResReal = cutComparefe(afRealRef, h_afReal, iSz, fEpsilon);
		printf("Checking %s result: %s\n", pcFileName, (cutbResReal) ? "PASSED" : "FAILED");
		//generate text file
#ifdef GENTXTOUTPUT
		FILE* fTxt = fopen(strcat(getFullPathOfFile(pcFileName), ".txt"), "w");
		for (int i = 0; i < iSz; i++)
		{
			fprintf(fTxt, "i = %d, My real: %f Ref real: %f\n", i, h_afReal[i], afRealRef[i]);
		}
		fclose(fTxt);
#endif
		//clean up memory
		if (!bHost) { free(h_afReal); }
		free(afRealRef);
	}
	fclose(fRef);

}

CompFlt_struct_t readCompFlt()
{
	CompFlt_struct_t gstCompFlt;
	FILE *fCompFlts = fopen(getFullPathOfFile("CompFlts.bin"), "rb");
	fread(&gstCompFlt.iH, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iW, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumIPRot, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumSz, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumOrigFlt, sizeof(int), 1, fCompFlts);
	fread(&gstCompFlt.iNumMulCompFlt, sizeof(int), 1, fCompFlts);
	int iNumTpl = gstCompFlt.iNumOrigFlt - gstCompFlt.iNumMulCompFlt;
	int iNumIPRotMemSz = gstCompFlt.iNumIPRot * sizeof(int);
	int iNumSzMemSz = gstCompFlt.iNumSz * sizeof(int);
	int iNumTplMemSz = iNumTpl * sizeof(int);
	int iNumAccResMemSz = iNumTpl * sizeof(AccRes_struct_t);
	gstCompFlt.iDataSz = gstCompFlt.iH * gstCompFlt.iW * gstCompFlt.iNumIPRot * gstCompFlt.iNumSz * gstCompFlt.iNumOrigFlt;
	gstCompFlt.iDataMemSz = gstCompFlt.iDataSz * sizeof(float);
#ifdef PINNED_MEM
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiIPAngs, iNumIPRotMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiTplCols, iNumSzMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.aiTpl_no, iNumTplMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gstCompFlt.h_afData, gstCompFlt.iDataMemSz));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gastAccRes, iNumAccResMemSz));
#else
	gstCompFlt.aiIPAngs = (int *)malloc(iNumIPRotMemSz);
	gstCompFlt.aiTplCols = (int *)malloc(iNumSzMemSz);
	gstCompFlt.aiTpl_no = (int *)malloc(iNumTplMemSz);
	gstCompFlt.h_afData = (float *)malloc(gstCompFlt.iDataMemSz);
	gastAccRes = (float *)malloc(iNumAccResMemSz);
#endif
	fread(gstCompFlt.aiIPAngs, sizeof(int), gstCompFlt.iNumIPRot, fCompFlts);
	fread(gstCompFlt.aiTplCols, sizeof(int), gstCompFlt.iNumSz, fCompFlts);
	fread(gstCompFlt.aiTpl_no, sizeof(int), iNumTpl, fCompFlts);

	fread(gstCompFlt.h_afData, sizeof(float), gstCompFlt.iDataSz, fCompFlts);
	fclose(fCompFlts);
	//initialized the accpsr to zero
	memset(gastAccRes, '\0', iNumAccResMemSz);
	return gstCompFlt;
}
void getKernelDims(int iBlockDimX, int iSz, dim3* dThreads, dim3* dBlocks)
{
	(*dThreads).x = iBlockDimX;
	int iGDx = (iSz) % (iBlockDimX) > 0 ? ((iSz) / (iBlockDimX)) + 1 : (iSz) / (iBlockDimX);
	(*dBlocks).x = iGDx;
	return;
}

//check if we will get the same results as Matlab FFT with using CUDA FFT by using a very small image
/*void cmpTest()
{
	hipfftReal *h_afTestReal, *h_afTestTpl, *d_afPadTestInReal;
	hipfftComplex *h_afTestCmplx, *d_afPadTestInCmplx, *d_afTestTplIn, *d_afTestTplOut, *d_afTestMul, *d_afTestCorr, *d_afPadTestOut_Real, *d_afPadTestOutCmplx;
	hipfftHandle hTestFFTplanFwdReal;
	hipfftHandle hTestFFTplanCmplx;
	hipfftHandle hTestFFTplanInvReal;
	float afTestSize[2];
	FILE *fTestSize = fopen(getFullPathOfFile("TestSize.bin"), "rb");
	fread(afTestSize, sizeof(float), 2, fTestSize);
	fclose(fTestSize);
	int iH = (int)afTestSize[0];
	int iW = (int)afTestSize[1];
	int iSz = iH*iW;
	int iMemSzReal = iSz * sizeof(hipfftReal);
	int iMemSzCmplx = iSz * sizeof(hipfftComplex);
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_afTestReal, iMemSzReal));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_afTestTpl, iMemSzReal));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_afTestCmplx, iMemSzCmplx));
	FILE *fTest = fopen(getFullPathOfFile("TestOrig.bin"), "rb");
	fread(h_afTestReal, sizeof(float), iSz, fTest);
	fclose(fTest);
	FILE *fTestTpl = fopen(getFullPathOfFile("TestTpl.bin"), "rb");
	fread(h_afTestTpl, sizeof(float), iSz, fTestTpl);
	fclose(fTestTpl);
	for (int iI = 0; iI<iSz; iI++)
	{
		h_afTestCmplx[iI].x = h_afTestReal[iI];
		h_afTestCmplx[iI].y = 0;
	}

	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTestInReal, iMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afTestTplIn, iMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTestInCmplx, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTestOutCmplx, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTestOut_Real, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afTestTplOut, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afTestMul, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afTestCorr, iMemSzCmplx));
	CUDA_SAFE_CALL(hipMemcpy(d_afPadTestInReal, h_afTestReal, iMemSzReal, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_afTestTplIn, h_afTestTpl, iMemSzReal, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_afPadTestInCmplx, h_afTestCmplx, iMemSzCmplx, hipMemcpyHostToDevice));
	CUFFT_SAFE_CALL(hipfftPlan2d(&hTestFFTplanFwdReal, iH, iW, HIPFFT_R2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&hTestFFTplanCmplx, iH, iW, HIPFFT_C2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&hTestFFTplanInvReal, iH, iW, HIPFFT_C2R));
	CUFFT_SAFE_CALL(hipfftExecR2C(hTestFFTplanFwdReal, (hipfftReal *)d_afPadTestInReal, (hipfftComplex *)d_afPadTestOut_Real));
	CUFFT_SAFE_CALL(hipfftExecR2C(hTestFFTplanFwdReal, (hipfftReal *)d_afTestTplIn, (hipfftComplex *)d_afTestTplOut));
	CUFFT_SAFE_CALL(hipfftExecC2C(hTestFFTplanCmplx, (hipfftComplex *)d_afPadTestInCmplx, (hipfftComplex *)d_afPadTestOutCmplx, HIPFFT_FORWARD));

	//cmpCPU(d_afPadTestOut_Real, "TestFFT.bin", 1, iSz);
	//cmpCPU(d_afPadTestOutCmplx, "TestFFT.bin", 1, iSz);

	dim3 dThreads(1, 1, 1);
	dim3 dBlocks(1, 1);
	getKernelDims(BLOCKDIMX, iSz, &dThreads, &dBlocks);
	pointWiseMul << <dBlocks, dThreads >> >(d_afTestMul, d_afPadTestOut_Real, d_afTestTplOut, iSz, 1.0f / (float)iSz);
	CUFFT_SAFE_CALL(hipfftExecC2R(hTestFFTplanInvReal, (hipfftComplex *)d_afTestMul, (hipfftReal *)d_afTestCorr));
	cmpCPU(d_afTestCorr, "TestCorr.bin", 0, iSz, 0, (float)1e-6);

	ComplexScale << <32, 256 >> >(d_afPadTestOutCmplx, iSz, 1.0f / (float)iSz);
	ComplexScale << <32, 256 >> >(d_afPadTestOut_Real, iSz, 1.0f / (float)iSz);
	CUFFT_SAFE_CALL(hipfftExecC2R(hTestFFTplanInvReal, (hipfftComplex *)d_afPadTestOut_Real, (hipfftReal *)d_afPadTestInReal));
	CUFFT_SAFE_CALL(hipfftExecC2C(hTestFFTplanCmplx, (hipfftComplex *)d_afPadTestOutCmplx, (hipfftComplex *)d_afPadTestInCmplx, HIPFFT_BACKWARD));

	cmpCPU(d_afPadTestInReal, "TestFFTInvReal.bin", 0, iSz, 0, (float)1e-6);
	cmpCPU(d_afPadTestInCmplx, "TestFFTInvCmplx.bin", 1, iSz, 0, (float)1e-6);

	CUDA_SAFE_CALL(hipFree(d_afPadTestInReal));
	CUDA_SAFE_CALL(hipFree(d_afPadTestInCmplx));
	CUDA_SAFE_CALL(hipFree(d_afPadTestOut_Real));
	CUDA_SAFE_CALL(hipFree(d_afPadTestOutCmplx));
	CUDA_SAFE_CALL(hipFree(d_afTestTplIn));
	CUDA_SAFE_CALL(hipFree(d_afTestTplOut));
	CUDA_SAFE_CALL(hipFree(d_afTestCorr));
	CUDA_SAFE_CALL(hipFree(d_afTestMul));
	hipHostFree(h_afTestReal);
	hipHostFree(h_afTestTpl);
	hipHostFree(h_afTestCmplx);
	CUFFT_SAFE_CALL(hipfftDestroy(hTestFFTplanFwdReal));
	CUFFT_SAFE_CALL(hipfftDestroy(hTestFFTplanCmplx));
	CUFFT_SAFE_CALL(hipfftDestroy(hTestFFTplanInvReal));
	return;
}*/

inline void InitKerTim(int iSz)
{
#ifdef KERTIM
	if (iSz == giTplSz)
	{
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUT_SAFE_CALL(cutResetTimer(guiKerTim));
		CUT_SAFE_CALL(cutStartTimer(guiKerTim));
	}
#endif
}

inline void WrapKerTim(char* sKerName, int iSz)
{
#ifdef KERTIM
	if (iSz == giTplSz) //1(copyscn convert fix), 2(1stPassInit), 3(2ndPassInit), giScnSz (1stLoop), giTplSz(2ndLoop)
	{
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUT_SAFE_CALL(cutStopTimer(guiKerTim));
		double dTime = sdkGetTimerValue(guiKerTim);
		printf("%s time: %f msecs.\n", sKerName, dTime);
		g_dTotalKerTime += dTime;
	}
#endif
}

void MaxIdx(hipfftReal* d_afData, int iSz, int** d_piMaxIdx)
{
	int iGDx;
	if (iSz == giScnSzPad)// fist pass
		iGDx = giWholeMaxGDx; // = (640*480/512*8) = (307200/4096) = 75 - will need two passes 
	else
		iGDx = giPartMaxGDx;; // if TplSz = 60, (60*60/512*8)+1 = (3600/4096)+1 = 1 - will only need one pass 
							  //if TplSz is larger it is possible that we need two passes.
							  //gd_afBlockMaxs have enough storage for finding max in whole scene.
							  //so it is definitely enough for finding max in part scene

							  //max will do 2 passes. In the first pass there will be several blocks. 
							  //In the second	there will be only one block.

							  //now do the first pass: each thread will read EACHTHREADREADS pixels. 
							  //Each block reads BLOCKDIMX_MAX*EACHTHREADREADS = 512*8 = 4096 pixels

	dim3 thread(BLOCKDIMX_MAX, 1, 1);
	dim3 grid(iGDx, 1);

	//calculate block maxs
	InitKerTim(iSz);
	max_k << < grid, thread >> >(d_afData, NULL, iSz, gd_afBlockMaxs, gd_aiBlockMaxIdxs);
	WrapKerTim("Max1stPass", iSz);
	CUT_CHECK_ERROR("Kernel execution failed");

	if (iGDx == 1)
	{
		*d_piMaxIdx = gd_aiBlockMaxIdxs;
	}
	else
	{   //find golbal max form iGDx local max
		//now do the second pass: each thread will read EACHTHREADREADS blockmaxs. 
		//We have only one block and this block reads iGDx blockmaxs.
		//note that (iGDx/EACHTHREADREADS) <= BLOCKDIMX_MAX
		dim3 thread2(BLOCKDIMX_MAX, 1, 1);
		dim3 grid2(1, 1);

		// execute the kernel
		//calculate maxs of block maxs
		InitKerTim(iSz);
		max_k << < grid2, thread2 >> >(gd_afBlockMaxs, gd_aiBlockMaxIdxs, iGDx, gd_pfMax, gd_piMaxIdx);
		WrapKerTim("Max2ndPass", iSz);
		*d_piMaxIdx = gd_piMaxIdx;
		CUT_CHECK_ERROR("Kernel execution failed");
	}
}

//compute PSR value
float getPSR(hipfftReal* gd_afCorr, hipfftReal* gh_afArea, int* iPeakIndx, int iSz, int iW, int iH)
{
	int iI;
	int *d_piMaxIdx = NULL;
	MaxIdx(gd_afCorr, iSz, &d_piMaxIdx);
	InitKerTim(iSz);
	CUDA_SAFE_CALL(hipMemcpy(iPeakIndx, (int*)d_piMaxIdx, sizeof(int), hipMemcpyDeviceToHost));
	WrapKerTim("MemcpyD2HPeak", iSz);
	//find PSR on the cpu, because we are dealing with at most giAreaH x giAreaH elements
	int iMaxRow, iMaxCol;
	Indx2Coord(iW, *iPeakIndx, &iMaxRow, &iMaxCol);
	//The int4 type is a CUDA built-in type with four fields: x(RowBeg),y(RowEnd),z(ColBeg),w(ColEnd)
	int4 aiAreaCoord = getSurrCoord(iMaxRow, iMaxCol, giAreaH, iW, iH);// reutrn correct coord, cut boundary
	int iStart = (aiAreaCoord.x*iW) + aiAreaCoord.z;
	//area is not always giAreaH x giAreaH, it might be cut if the peak is close to boundary
	int iNewAreaH = aiAreaCoord.y - aiAreaCoord.x + 1;
	int iNewAreaW = aiAreaCoord.w - aiAreaCoord.z + 1;
	int iNewAreaSz = iNewAreaW*iNewAreaH;
	//transfer the data(Corr-ed) from device 
	InitKerTim(iSz);
	CUDA_SAFE_CALL(hipMemcpy2D(gh_afArea, iNewAreaW * sizeof(hipfftReal), gd_afCorr + iStart, iW * sizeof(hipfftReal), iNewAreaW * sizeof(hipfftReal), iNewAreaH, hipMemcpyDeviceToHost));
	WrapKerTim("MemcpyD2HArea", iSz);
	//find the new index of the max value in the area cut from corr plane
	float fMax = gh_afArea[0];
	int iNewMaxIndx = 0;
	for (iI = 0; iI<iNewAreaSz; iI++)
	{
		if (gh_afArea[iI] > fMax)
		{
			fMax = gh_afArea[iI];
			iNewMaxIndx = iI;
		}
	}
	int iNewMaxRow, iNewMaxCol;
	Indx2Coord(iNewAreaW, iNewMaxIndx, &iNewMaxRow, &iNewMaxCol);
	int4 aiMaskCoord = getSurrCoord(iNewMaxRow, iNewMaxCol, giMaskH, iNewAreaW, iNewAreaH);
	//mask is not always giMaskH x giMaskH, it might be cut if the peak is close to boundary
	int iNewMaskH = aiMaskCoord.y - aiMaskCoord.x + 1;
	int iNewMaskW = aiMaskCoord.w - aiMaskCoord.z + 1;
	//assign mask values to zero
	assignVal(iNewAreaW, gh_afArea, aiMaskCoord, 0);
	//calculate mean by not counting the mask
	int iFrameNumElem = (iNewAreaH*iNewAreaW) - (iNewMaskH*iNewMaskW);
	float fMean = sum(gh_afArea, iNewAreaSz) / iFrameNumElem;
	//mask values = mean
	assignVal(iNewAreaW, gh_afArea, aiMaskCoord, fMean);
	//calculate standard deviation by not counting the mask
	//calculate sum of sqr_dif
	float fTotal = 0;
	float fVal;
	for (iI = 0; iI < iNewAreaSz; iI++)
	{
		fVal = gh_afArea[iI] - fMean;
		fTotal += fVal*fVal;
	}
	float afStdVar = sqrt(fTotal / (iFrameNumElem - 1));
	float fMeasure;
	if (afStdVar != 0)
		fMeasure = (fMax - fMean) / afStdVar;
	else
		//if we are out of bound while copying part scene, this might happen since part scene will have lots of zeros
		fMeasure = 0;
	return fMeasure;
}

void Corr(hipfftComplex* d_afTplOut, dim3 dBlocks, dim3 dThreads, hipfftComplex* d_afScnOut, int iSz, hipfftComplex* gd_afMul, hipfftHandle hFFTplanInv, hipfftReal* gd_afCorr, hipfftReal* gh_afArea, int* piPeakIndx, float* pfPSR, int iW, int iH)
{

	//take conjugate of template fft and point wise multiply with scene and scale it with image size
	InitKerTim(iSz);
	pointWiseMul << <dBlocks, dThreads >> >(gd_afMul, d_afScnOut, d_afTplOut, iSz, 1.0f / (float)iSz);//gd_afMul is the output of the kernel
	WrapKerTim("Mul", iSz);
	CUT_CHECK_ERROR("pointWiseMul() execution failed\n");
	//take inverse FFT of multiplication
	InitKerTim(iSz);
	CUFFT_SAFE_CALL(hipfftExecC2R(hFFTplanInv, (hipfftComplex *)gd_afMul, (hipfftReal *)gd_afCorr));
	WrapKerTim("FFTinv", iSz);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	//find the PSR
	*pfPSR = getPSR(gd_afCorr, gh_afArea, piPeakIndx, iSz, iW, iH);// return *piPeakIndex and fMeasure
	return;
}
inline void InitTim()
{
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkResetTimer(guiParTim));
	CUT_SAFE_CALL(sdkStartTimer(guiParTim));
#endif
}

inline void WrapTim(char* sParName)
{
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkStopTimer(guiParTim));
	double dTime = sdkGetTimerValue(guiParTim);
	printf("%s time: %f msecs.\n", sParName, dTime);
	g_dRunsOnGPUTotalTime += dTime;
#endif
}

void PrepTplFFT(hipfftReal* gd_afCompFlt, hipfftReal** d_pafPadTplIn, hipfftComplex** d_pafPadTplOut, hipfftComplex** d_pafWholeTplFFT, hipfftComplex** d_pafPartTplFFT, hipfftHandle ghFFTplanWholeFwd, hipfftHandle ghFFTplanPartFwd)
{
#ifdef SAVEFFT
	int iSzIndx, iIPIndx, iFltIndx, iFltAbsIndx, iIPAbsIndx;
	hipfftReal
		*d_afTpl,
		*d_afPadTplIn;
	//first allocate mem
	//WholeTpls are the MulCompFlts (last flts in the compflt list). They are used in 1st pass. Their size is as big as scn
	//PartTpls are all other comp flt excluding MulCompFlts. They are used in 2nd pass. Their size is as big as tpl (is not blowed up to scn size)
	int iWholeMemSz = giScnHPad * giScnWPad * giNumIPInFirst * giNumSz * gstCompFlt.iNumMulCompFlt * sizeof(hipfftComplex);
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafWholeTplFFT, iWholeMemSz));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_afPadTplIn, giScnMemSzRealPad));
	int iPartMemSz = giTplH * giTplW * giNumIPRot * giNumSz * giNumSngCompFlt * sizeof(hipfftComplex);
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPartTplFFT, iPartMemSz));
	//take FFT of WholeTpls
	for (iFltIndx = giNumSngCompFlt; iFltIndx < giNumOrigFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = giBegIdxIPInFirst; iIPIndx < giEndIdxIPInFirst; iIPIndx++)
			{
				CUDA_SAFE_CALL(hipMemset(d_afPadTplIn, 0, giScnMemSzRealPad));
				d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
				//pad template
				CUDA_SAFE_CALL(hipMemcpy2D(d_afPadTplIn, (giScnWPad * sizeof(hipfftReal)), d_afTpl, giTplWMemSz, giTplWMemSz, giTplH, hipMemcpyDeviceToDevice));
				//take the fft and save it to WholeTplFFT
				iFltAbsIndx = iFltIndx - giNumSngCompFlt;
				iIPAbsIndx = iIPIndx - giBegIdxIPInFirst;
				printf("iIPIndx=%d iSzIndx=%d iFltIndx=%d d_afPadTplIn= %d\n", iIPIndx, iSzIndx, iFltIndx, d_afPadTplIn);
				CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)d_afPadTplIn, (hipfftComplex *)*d_pafWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx)));

			}
		}
	}
	CUDA_SAFE_CALL(hipFree(d_afPadTplIn));
	//take FFT of PartTpls
	for (iFltIndx = 0; iFltIndx < giNumSngCompFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = 0; iIPIndx < giNumIPRot; iIPIndx++)
			{
				d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
				printf("iIPIndx=%d iSzIndx=%d iFltIndx=%d d_afTpl= %d\n", iIPIndx, iSzIndx, iFltIndx, d_afTpl);
				//take the fft and save it to PartTplFFT
				CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)d_afTpl, (hipfftComplex *)*d_pafPartTplFFT(iIPIndx, iSzIndx, iFltIndx)));
			}
		}
	}
#else
	//allocate gd_afPadTplIn and gd_afPadTplOut 
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPadTplIn, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMemset(*d_pafPadTplIn, 0, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&*d_pafPadTplOut, giScnMemSzCmplx));
#endif
}

void DestroyTplFFT(hipfftComplex* gd_afWholeTplFFT, hipfftComplex* gd_afPartTplFFT, hipfftReal* gd_afPadTplIn, hipfftComplex* gd_afPadTplOut)
{
#ifdef SAVEFFT
	CUDA_SAFE_CALL(hipFree(gd_afWholeTplFFT));
	CUDA_SAFE_CALL(hipFree(gd_afPartTplFFT));
#else
	CUDA_SAFE_CALL(hipFree(gd_afPadTplIn));
	CUDA_SAFE_CALL(hipFree(gd_afPadTplOut));
#endif
}

void getWholeTplFFT(hipfftReal* gd_afCompFlt, int iIPIndx, int iSzIndx, int iFltIndx, hipfftReal* gd_afPadTplIn, hipfftComplex** d_pafPadTplOut, hipfftHandle ghFFTplanWholeFwd, hipfftComplex* gd_afWholeTplFFT)
{
#ifdef SAVEFFT
	int iFltAbsIndx = iFltIndx - giNumSngCompFlt;
	int iIPAbsIndx = iIPIndx - giBegIdxIPInFirst;
	*d_pafPadTplOut = gd_afWholeTplFFT(iIPAbsIndx, iSzIndx, iFltAbsIndx);
#else
	//find the starting index of template
	hipfftReal* d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
	//pad template
	CUDA_SAFE_CALL(hipMemcpy2D(gd_afPadTplIn, (giScnW * sizeof(hipfftReal)), d_afTpl, giTplWMemSz, giTplWMemSz, giTplH, hipMemcpyDeviceToDevice));
	//take the FFT of the template
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)gd_afPadTplIn, (hipfftComplex *)*d_pafPadTplOut));
#endif
}

void getPartTplFFT(hipfftReal* gd_afCompFlt, int iIPIndx, int iSzIndx, int iFltIndx, hipfftComplex** d_pafPadTplOut, hipfftHandle ghFFTplanPartFwd, hipfftComplex* gd_afPartTplFFT)
{
#ifdef SAVEFFT
	*d_pafPadTplOut = gd_afPartTplFFT(iIPIndx, iSzIndx, iFltIndx);
#else
	//get the pointer to the tpl
	hipfftReal* d_afTpl = gd_afCompFlt(iIPIndx, iSzIndx, iFltIndx);
	//take the FFT of the template
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)d_afTpl, (hipfftComplex *)*d_pafPadTplOut));
#endif
}

//If the MaxPeakIndex is close to the boundry of the image, when we try to copy the part of the scene
//we can be out bound! check for this condition, and if so get part of the scene until boundry.
//since we initalize the part image to zero, it would have an effect such that part image is padded with zero
void getCopyWidthHeight(int iMaxPeakIndx, int* piPartW, int* piPartH)
{
	int iMaxPeakRow, iMaxPeakCol;
	//make sure we are not out of bounds
	Indx2Coord(giScnWPad, iMaxPeakIndx, &iMaxPeakRow, &iMaxPeakCol);// converted coord are the same with 640*480 and 1024*1024
	*piPartW = giTplW;
	int iEndCol = iMaxPeakCol + *piPartW - 1;
	if (iEndCol >= giScnW && iMaxPeakCol <= giScnW)
		*piPartW = *piPartW - (iEndCol + 1 - giScnW);
	*piPartH = giTplH;
	int iEndRow = iMaxPeakRow + *piPartH - 1;
	if (iEndRow >= giScnH && iMaxPeakRow <= giScnW)
		*piPartH = *piPartH - (iEndRow + 1 - giScnH);
}

/*B = GRAYTO8(A) converts the double array A to unisgned char by scaling A by 255
* and then rounding.  NaN's in A are converted to 0.  Values in A greater
* than 1.0 are converted to 255; values less than 0.0 are converted to 0.
*/
void ConvertFromDouble(float *pr, unsigned char *qr, int numElements)
{
	int k;
	float val;

	for (k = 0; k < numElements; k++)
	{
		val = *pr++;
		if (val == NULL) {
			*qr++ = 0;
		}
		else {
			val = val * 255.0f + 0.5f;
			if (val > 255.0) val = 255.0;
			if (val < 0.0)   val = 0.0;
			*qr++ = (unsigned char)val;
		}
	}
}

//this function immitates Matlab imadjust function's LookUp Table creation.
void genLUT()
{
	float fN = LUTSIZE;
	float fD1 = 0;
	float fD2 = 1;
	for (int i = 0; i < fN - 1; i++)
	{
		gfLUT[i] = fD1 + i*((fD2 - fD1) / (fN - 1));
	}
	gfLUT[int(fN - 1)] = fD2;

	//make sure lut is in the range [gfLIn;gfHIn]
	for (int i = 0; i < fN; i++)
	{
		if (gfLUT[i] < gfLIn) gfLUT[i] = gfLIn;
		if (gfLUT[i] > gfHIn) gfLUT[i] = gfHIn;
	}

	//out = ( (img - lIn(d,:)) ./ (hIn(d,:) - lIn(d,:)) ) .^ (g(d,:));
	for (int i = 0; i < fN; i++)
	{
		gfLUT[i] = pow((gfLUT[i] - gfLIn) / (gfHIn - gfLIn), gfG);
	}
	//out(:) = out .* (hOut(d,:) - lOut(d,:)) + lOut(d,:);
	for (int i = 0; i < fN; i++)
	{
		gfLUT[i] = gfLUT[i] * (gfHOut - gfLOut) + gfLOut;
	}
	ConvertFromDouble(gfLUT, gacLUT, LUTSIZE);
}

void CpyScnToDevAndPreProcess(unsigned char* acScn, float* d_afPadScnIn, bool bConGam, bool bFixDead)
{
	//I can do the adjusting before fixing the dead pixel. Adjusted dead pixel will be overwritten as an overage of adjusted neighbors. Adjusting is done to each pixel independently.
	//copy scene to device
	InitTim();
	InitKerTim(1);
	CUDA_SAFE_CALL(hipMemcpy(gd_ac4Scn, acScn + giScnOffset, giScnMemSzUChar, hipMemcpyHostToDevice));
	WrapKerTim("MemcpyH2DScn", 1);
	WrapTim("CopyFrameToGPUMem");

	InitTim();
	InitKerTim(1);
	convertChar4ToFloatDoConGam << <gdBlocksConv, gdThreadsConv >> > (gd_ac4Scn, (float4*)d_afPadScnIn, (giScnSz / 4), bConGam);
	WrapKerTim("ConvertScn", 1);
	WrapTim("convertChar4ToFloatDoConGam");

	if (bFixDead)
	{
		InitTim();
		InitKerTim(1);
		fixDeadPixels << <gdBlocksDead, gdThreadsDead >> > ((hipfftReal*)d_afPadScnIn, giScnSz, giScnW, giScnH);
		WrapKerTim("FixScn", 1);
		WrapTim("fixDeadPixel");
	}

#ifdef COPYBACKAFTERDEADFIX
	//only for visualization purposes. no need to optimize below code with kernels.
	/*hipfftReal* h_afScnOut = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScnOut, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	for (int i = 0; i < giScnSz; i++)
		acScn[i + giScnOffset] = (unsigned char)h_afScnOut[i];
	free(h_afScnOut);*/
#endif

#ifdef SAVEFIXEDSCN
	hipfftReal* h_afScn = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScn, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	FILE *fFixedScn = fopen(getFullPathOfFile("fixedScn.bin"), "wb");
	fwrite(h_afScn, sizeof(hipfftReal), giScnSz, fFixedScn);
	fclose(fFixedScn);
	free(h_afScn);
#endif

#ifdef RUNCLAHE
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkResetTimer(guiParTim));
	CUT_SAFE_CALL(sdkStartTimer(guiParTim));
#endif
	//IMPLEMENT THIS SECTION ON GPU: only for testing CLAHE it is running on the CPU
	hipfftReal* h_afScnClahe = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScnClahe, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	unsigned char* acScnClahe = (unsigned char*)malloc(giScnMemSzUChar);
	for (int i = 0; i < giScnSz; i++)
		{acScnClahe[i] = (unsigned char)h_afScnClahe[i];}
	//convert to unsigned int
	CLAHE(acScnClahe, giScnW, giScnH, 0, 255, giScnW / 8, giScnH / 8, 256, 0.3f); //80 60, 80 30
																				  //copy scene to device
	CUDA_SAFE_CALL(hipMemcpy(gd_ac4Scn, acScnClahe, giScnMemSzUChar, hipMemcpyHostToDevice));
	convertChar4ToFloatDoConGam << <gdBlocksConv, gdThreadsConv >> >(gd_ac4Scn, (float4*)d_afPadScnIn, (giScnSz / 4), bConGam);
	free(h_afScnClahe);
	free(acScnClahe);
#ifdef PARTIM
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(sdkStopTimer(guiParTim));
	g_dClaheTime = sdkGetTimerValue(guiParTim);
	printf("Clahe (runs on the CPU!) time: %f msecs.\n", g_dClaheTime);
#endif
	////////////////////
#endif

#ifdef COPYBACK
	//only for visualization purposes. no need to optimize below code with kernels.
	hipfftReal* h_afScnOut = (hipfftReal*)malloc(giScnMemSzReal);
	CUDA_SAFE_CALL(hipMemcpy(h_afScnOut, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	for (int i = 0; i<giScnSz; i++)
		acScn[i + giScnOffset] = (unsigned char)h_afScnOut[i];
	free(h_afScnOut);
#endif
	//added to show CLAHE effect in the ssd_fft_gpu_GUI
	if (giShowClaheGUI == -1)
	{
		//only for visualization purposes. no need to optimize below code with kernels.
		hipfftReal* h_afScnOutGUI = (hipfftReal*)malloc(giScnMemSzReal);
		CUDA_SAFE_CALL(hipMemcpy(h_afScnOutGUI, d_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
		for (int i = 0; i<giScnSz; i++)
			acScn[i + giScnOffset] = (unsigned char)h_afScnOutGUI[i];
		free(h_afScnOutGUI);
	}
}


void DisplayResults(float fPSR, int iTplIndx, int iIPIndx, int iSzIndx, int iStatsFrameCur)
{
	giSLCurFrm = -1;
	giSLResult = -1;
#ifdef DISP_FRM_RECOG
	if (fPSR > gfPSRTrashold)
	{
		//printf("Max PSR value: %f (TplNo = %d, IPAng = %d, Sz = %d)\n", fPSR, gstCompFlt.aiTpl_no[iTplIndx], gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
		printf("Frame votes for %3d %s (PSR: %5.2f, in-plane rotation: %3d\xf8, size: %2d)\n", gstCompFlt.aiTpl_no[iTplIndx], acMeasure, fPSR, gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
		giSLCurFrm = gstCompFlt.aiTpl_no[iTplIndx];
	}
	//	else
	//		printf("\n");
#endif

#ifdef MAJVOT
	int iNumTpl = gstCompFlt.iNumOrigFlt - gstCompFlt.iNumMulCompFlt;
	float fAddConfIP, fAddConfSz;

	//update the AccRes
	if (giFrameNo == 0)
	{
		if (fPSR > gfPSRTrashold)
		{
			//start the tracking at the first seen sign
			giFrameNo++;
			gastAccRes[iTplIndx].fAccConf = gastAccRes[iTplIndx].fAccConf + fPSR;
			giNumFramesInAcc++;
			gastAccRes[iTplIndx].iPrevIP = iIPIndx;
			gastAccRes[iTplIndx].iPrevSz = iSzIndx;
		}
	}
	else
	{
		//increase the tracked frameNum regardless of the PSR value if we already started the tracking
		giFrameNo++;
		if (fPSR > gfPSRTrashold)
		{
			fAddConfIP = 0;
			fAddConfSz = 0;
			if (gastAccRes[iTplIndx].fAccConf > 0)
			{
				//there has been a previous recognition of this tpl (iPrevIP and iPrevSz has valid values)
				//increase confidence if IP is the same as previous and/or Sz is getting larger.
				if ((iIPIndx - gastAccRes[iTplIndx].iPrevIP) == 0)
					fAddConfIP = gfAddConfIPFac*fPSR;
				if ((iSzIndx - gastAccRes[iTplIndx].iPrevSz) == 0)
					fAddConfSz = gfAddConfEqSzFac*fPSR;
				else if ((iSzIndx - gastAccRes[iTplIndx].iPrevSz) > 0)
					fAddConfSz = gfAddConfGrSzFac*fPSR;
			}
			gastAccRes[iTplIndx].fAccConf = gastAccRes[iTplIndx].fAccConf + fPSR + fAddConfIP + fAddConfSz;
			giNumFramesInAcc++;
			gastAccRes[iTplIndx].iPrevIP = iIPIndx;
			gastAccRes[iTplIndx].iPrevSz = iSzIndx;
		}
	}

	int iMaxTplIndx = -1;
	if (giFrameNo == giTrackingLen)
	{
		//find the bestTpl
		float fMaxAccConf = gastAccRes[0].fAccConf;
		iMaxTplIndx = 0;
		for (int i = 1; i<iNumTpl; i++)
		{
			if (gastAccRes[i].fAccConf > fMaxAccConf)
			{
				iMaxTplIndx = i;
				fMaxAccConf = gastAccRes[i].fAccConf;
			}
		}
		//printf("\n           Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
#ifdef REALTIME
		if (fMaxAccConf > gfAccPSRTrasholdSpecialReal && giNumFramesInAcc == 1 && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2)
			printf("\n           Best Tpl = %d (Max AccConf = %f)\n(special rule for realtime emulation=> result is based on only ONE frame with VERY high confidence)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
		else if (fMaxAccConf > gfAccPSRTrashold && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2) //2 = 00t
			printf("\n           Best Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
		else
			iMaxTplIndx = -1;
#else
		if (fMaxAccConf > gfAccPSRTrashold && gstCompFlt.aiTpl_no[iMaxTplIndx] != 2) //2 = 00t
		{
			//printf("\n           Best Tpl = %d (Max AccConf = %f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], fMaxAccConf);
			printf("\n      System concludes that speed limit is %3d %s! (Total votes: %6.2f)\n\n", gstCompFlt.aiTpl_no[iMaxTplIndx], acMeasure, fMaxAccConf);
			giSLResult = gstCompFlt.aiTpl_no[iMaxTplIndx];
		}
		else
			iMaxTplIndx = -1;
#endif
		giFrameNo = 0;
		giNumFramesInAcc = 0;
		//initialize the accpsr to zero
		memset(gastAccRes, '\0', (iNumTpl * sizeof(AccRes_struct_t)));
	}

#ifdef STATS
	// Print the best sign found in the sequence of frames to the stats file
	if (iMaxTplIndx != -1)
		fprintf(g_fStatsFile, "%d\t%d\n", iStatsFrameCur, gstCompFlt.aiTpl_no[iMaxTplIndx]);
	else
		fprintf(g_fStatsFile, "%d\t0\n", iStatsFrameCur);
#endif
#endif
}

#ifndef US_SIGNS 
#ifdef STATS
void IncFPSCount(unsigned long ulTimeStamp, int iFrameCur)
{
	int iDiff = ulTimeStamp - g_ulLastTimeStamp;
	if (iDiff >= 61 && iDiff <= 64) //mostly 62, 63
		gi16fps++;
	else if (iDiff >= 123 && iDiff <= 126) //mostly 124, 125
		gi8fps++;
	else if (iDiff >= 185 && iDiff <= 188) //mostly 186, 187
		gi5fps++;
	else if (iDiff >= 247 && iDiff <= 250) //mostly 248, 249
		gi4fps++;
	else if (iDiff == 0)
		gi0fps++;
	else
		fprintf(g_fStatsFile, "%d\tTimeNotKnown\t%d\n", iFrameCur, iDiff);

}
#endif
#endif
////////////////////////////////////////////////////////////////////////////////
// Member Functions
////////////////////////////////////////////////////////////////////////////////
void ssd_fft_gpu_init()
{

#ifdef CMPTEST
	cmpTest();
#endif

#ifdef DISP_DEV_INIT
	//display the device info
	int iDeviceCount;
	hipGetDeviceCount(&iDeviceCount);
	printf("Device Count: %d\n", iDeviceCount);
	hipSetDevice(0); //when using animas since there is only 1 GPU (G80) ID must be 0 (otherwise it fails on fist fft execution), on burn we can set it to any ID we want
	int iDev;
	struct hipDeviceProp_t prop;
	hipGetDevice(&iDev);
	hipGetDeviceProperties(&prop, iDev);
	printf("The Properties of the Device with ID %d are\n", iDev);
	printf("\tDevice Name : %s\n", prop.name);
	printf("\tDevice Total Global Memory Size (MBs) : %u\n", prop.totalGlobalMem / 1048576); //1 Megabyte = 1048576 Bytes
	printf("\tDevice Total Constant Memory Size (KBs) : %u\n", prop.totalConstMem / 1024); //1 KB = 1024 Bytes
	printf("\tDevice # of MultiProcessors : %d\n", prop.multiProcessorCount); //1SM(streaming processor has 8 SM (streaming processors = cores)
#endif
#ifdef DISP_DEV_INIT
	printf("Read composite filters...\n");
#endif
	//read comp filters
	gstCompFlt = readCompFlt();
	giTplH = gstCompFlt.iH;//
	giTplW = gstCompFlt.iW;//
	printf("giTplW=%d\n", giTplW);
	printf("giTplH=%d\n", giTplH);
	giTplSz = giTplH * giTplW;
	giTplWMemSz = giTplW * sizeof(hipfftReal);
	giTplMemSzReal = giTplH * giTplW * sizeof(hipfftReal);
	giTplMemSzCmplx = giTplH * giTplW * sizeof(hipfftComplex);
	giNumIPRot = gstCompFlt.iNumIPRot;
	giNumSz = gstCompFlt.iNumSz;
	giNumOrigFlt = gstCompFlt.iNumOrigFlt;
	giNumSngCompFlt = giNumOrigFlt - gstCompFlt.iNumMulCompFlt;

	//do some check
	giPartMaxGDx = (giTplSz) % (BLOCKDIMX_MAX*EACHTHREADREADS) > 0 ? ((giTplSz) / (BLOCKDIMX_MAX*EACHTHREADREADS)) + 1 : (giTplSz) / (BLOCKDIMX_MAX*EACHTHREADREADS);
	if (giPartMaxGDx > 1)
	{
		printf("Warning: Max of part scn can not be found in one pass!\n");
	}
	giWholeMaxGDx = (giScnSzPad) % (BLOCKDIMX_MAX*EACHTHREADREADS) > 0 ? ((giScnSzPad) / (BLOCKDIMX_MAX*EACHTHREADREADS)) + 1 : (giScnSzPad) / (BLOCKDIMX_MAX*EACHTHREADREADS);
	if ((giWholeMaxGDx / EACHTHREADREADS) > BLOCKDIMX_MAX)
	{
		//in the second pass each thread will read EACHTHREADREADS blockmaxs. There is giWholeMaxGDx blocks at most.
		//if giWholeMaxGDx/EACHTHREADREADS > BLOCKDIMX_MAX this means that second pass should have more than one block.
		//but it should have only one!
		printf("Error: Each thread in max kernel should read more than %d elements!\n", EACHTHREADREADS);
		exit(0);
	}
#ifdef DISP_DEV_INIT
	printf("Allocating memory...\n");
#endif
#ifdef PINNED_MEM
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gh_acScn, giOrigScnMemSzUChar));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&gh_afArea, giAreaMemSzReal));
#else
	gh_acScn = (unsigned char *)malloc(giOrigScnMemSzUChar);
	gh_afArea = (hipfftReal *)malloc(giAreaMemSzReal);
#endif
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_ac4Scn, giScnMemSzUChar));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afCompFlt, gstCompFlt.iDataMemSz));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnIn, giScnMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnInPad, giScnMemSzRealPad));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afCorr, giScnMemSzRealPad));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnOut, giScnMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afPadScnOutPad, giScnMemSzCmplxPad));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afMul, giScnMemSzCmplxPad));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afScnPartIn, giTplMemSzReal));
	CUDA_SAFE_CALL(hipMalloc((void **)&gd_afScnPartOut, giTplMemSzCmplx));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_pfMax, sizeof(hipfftReal)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_afBlockMaxs, sizeof(hipfftReal)*giWholeMaxGDx));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_piMaxIdx, sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&gd_aiBlockMaxIdxs, sizeof(int)*giWholeMaxGDx));

	gd_afWholeTplFFT = NULL;
	gd_afPartTplFFT = NULL;
	gd_afPadTplIn = NULL;
	gd_afPadTplOut = NULL;

#ifdef DISP_DEV_INIT
	printf("Scene size      : %i x %i\n", giScnW, giScnH);
	printf("Template size	: %i x %i\n", giTplW, giTplH);
#endif

	//calculate the block and grid size (both will be 1D) to be used by kernels
	getKernelDims(BLOCKDIMX, giScnSz / 4, &gdThreadsConv, &gdBlocksConv);
	getKernelDims(BLOCKDIMX, giScnSz / 2, &gdThreadsDead, &gdBlocksDead);
	gdThreadsDead.x = gdThreadsDead.x + (HALFWARP + 1);
	getKernelDims(BLOCKDIMX, giScnSzPad, &gdThreadsWhole, &gdBlocksWhole);// for first pass
	getKernelDims(BLOCKDIMX, giTplSz, &gdThreadsPart, &gdBlocksPart);

	//Creating FFT plan for whole scene
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanWholeFwd, giScnHPad, giScnWPad, HIPFFT_R2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanWholeInv, giScnHPad, giScnWPad, HIPFFT_C2R));
	//Creating FFT plan for part of the scene
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanPartFwd, giTplH, giTplW, HIPFFT_R2C));
	CUFFT_SAFE_CALL(hipfftPlan2d(&ghFFTplanPartInv, giTplH, giTplW, HIPFFT_C2R));
	printf("%d %d", giTplH, giTplW);

	//CUT_SAFE_CALL( sdkCreateTimer(&guiParTim) );
	sdkCreateTimer(&guiParTim);
	//CUT_SAFE_CALL( sdkCreateTimer(&guiKerTim) );
	sdkCreateTimer(&guiKerTim);

	InitTim();
	//copy all Composite Filters to device memory (copying device to device would take less time)
	CUDA_SAFE_CALL(hipMemcpy(gd_afCompFlt, gstCompFlt.h_afData, gstCompFlt.iDataMemSz, hipMemcpyHostToDevice));
	//figure out params regarding IPRot
#ifdef DoIPInSecond
	giBegIdxIPInFirst = giNumIPRot / 2; //middle is the not-IProtated compFlt
	giEndIdxIPInFirst = giBegIdxIPInFirst + 1;
	giNumIPInFirst = 1;
	giBegIdxIPInSecond = 0;
	giEndIdxIPInSecond = giNumIPRot;
#else
	giBegIdxIPInFirst = 0;
	giEndIdxIPInFirst = giNumIPRot;
	giNumIPInFirst = giNumIPRot;
	//assign second pass params on-line
#endif
	PrepTplFFT(gd_afCompFlt, &gd_afPadTplIn, &gd_afPadTplOut, &gd_afWholeTplFFT, &gd_afPartTplFFT, ghFFTplanWholeFwd, ghFFTplanPartFwd);
	WrapTim("PrepTplFFT");
	if (gbConGam)
	{
		genLUT();
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_acLUT), gacLUT, sizeof(unsigned char)*LUTSIZE));
	}

#ifndef REALTIME
	gfAccPSRTrashold = 31.5f;//32;//before: gfAccPSRFac*gfPSRTrashold if clahelimit = 0.6f fac = 4, if clahelimit= 0.3f (less noise) fac = 5 to avoid FP, TN
#else
	gfAccPSRTrashold = 25.95f;
#endif
}

void BestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp)
{
	int iIPIndx, iSzIndx, iFltIndx, iPeakIndx;

	float fPSR, fMaxPSR;

	int iMaxIPIndx, iMaxSzIndx, iMaxFltIndx;

	int iPartWMemSz;

	giShowClaheGUI = file_info[2];
	//save the scn in bin file (to transfer the videos from Realis to GUI)
#ifdef SAVESCNBIN
	// If the first frameID
	if (file_info[0] == file_info[1])
	{
		if (g_fScnBin != NULL)
			fclose(g_fScnBin);
		char acScnName[] = "00000.txt";
		itoa(file_info[1], acScnName, 10);
		strcpy(g_sScnBinPath, g_sScnBinPathBegin);
		strcat(g_sScnBinPath, acScnName);
		strcat(g_sScnBinPath, ".bin\0");
		g_fScnBin = fopen(g_sScnBinPath, "wb");
	}
	fwrite(acScn, sizeof(unsigned char), giOrigScnSz, g_fScnBin);
#endif

#ifdef STATS
	// current Frame ID 
	int iFrameCur = file_info[0];
	// First Frame ID
	int iFrameBeg = file_info[1];
	// If the first frameID
	if (iFrameCur == iFrameBeg)
	{
		//close prev stats file 
		if (g_fStatsFile != NULL)
		{
			//close the stats file for prev video
			fclose(g_fStatsFile);
			//add this videos time to all video time
			g_iNumVideos++;
			g_fAllVideoTime = g_fAllVideoTime + (float)(((double)g_ulLastTimeStamp - (double)g_ulFirstTimeStamp) / 1000 / 60);
		}
		//start the time to calculate current video time
		g_ulFirstTimeStamp = ulTimeStamp;
		//open a stats file for current video
		strcpy(g_sStatsPath, g_sStatsPathBegin);
#ifndef US_SIGNS
		char acFName[] = "00000.txt";
		itoa(iFrameBeg, acFName, 10);
		strcat(g_sStatsPath, acFName);
#else
		strcat(g_sStatsPath, gacClipName);
#endif
		strcat(g_sStatsPath, ".txt\0");
		g_fStatsFile = fopen(g_sStatsPath, "wb");
		if (g_fStatsFile == NULL)
			printf("Error openning stats file!");
	}
	else //if not the first frame in the video, increment the appropriate FPS(frames per second) counter
	{
#ifndef US_SIGNS
		IncFPSCount(ulTimeStamp, iFrameCur);
#endif
	}
	g_ulLastTimeStamp = ulTimeStamp;
#endif

	*piMaxPeakIndx = -1;
	*piPartW = -1;
	*piPartH = -1;
#ifdef REALTIME
	int iTimeDiff;
	if (file_info[0] > file_info[1]) //if it is not the first frame 
	{
		iTimeDiff = ulTimeStamp - g_ulPrevTimeStamp;
		if (iTimeDiff < g_iRuntime) //do not process this frame
		{
#ifdef STATS
			fprintf(g_fStatsFile, "%d\t-1\n", iFrameCur); //enter -1 as speed sign found
#endif
			return;
		}
		else
			g_ulPrevTimeStamp = ulTimeStamp;
	}
	else
		g_ulPrevTimeStamp = ulTimeStamp;
#endif

	bool bLoadScn = false;
	//Read scene...
	if (acScn == NULL)
	{
		//no video input, process the scn from file
		FILE *fScn = fopen(getFullPathOfFile("scn.bin"), "rb");
		fread(gh_acScn, sizeof(unsigned char), giOrigScnSz, fScn);
		fclose(fScn);
		acScn = gh_acScn;
		bLoadScn = true;
	}
	/*	else
	{
	FILE *fScnIn = fopen(getFullPathOfFile("scnV.bin"), "wb");
	fwrite(acScn, sizeof(unsigned char), giOrigScnSz, fScnIn);
	fclose(fScnIn);
	FILE *fScn = fopen(getFullPathOfFile("scnV.bin"), "rb");
	fread(gh_acScn, sizeof(unsigned char), giOrigScnSz, fScn);
	fclose(fScn);
	acScn = gh_acScn;
	}
	*/
	bool bFixDead = gbFixDead;
	if (bLoadScn) bFixDead = 0;

	////////FIRST PASS///////////
#ifdef ALLTIM
	unsigned int uiAllTim;
	CUT_SAFE_CALL(cutCreateTimer(&uiAllTim));
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUT_SAFE_CALL(cutResetTimer(uiAllTim));
	CUT_SAFE_CALL(cutStartTimer(uiAllTim));
#endif
#ifdef PARTIM
	g_dRunsOnGPUTotalTime = 0;
#endif
#ifdef KERTIM
	g_dTotalKerTime = 0;
#endif

	CpyScnToDevAndPreProcess(acScn, gd_afPadScnIn, gbConGam, bFixDead);
	// pad gd_afPadScnIn to 1024*1024 gd_afPadScnInPad
	CUDA_SAFE_CALL(hipMemset(gd_afPadScnInPad, 0, giScnMemSzRealPad));
	//pad template
	CUDA_SAFE_CALL(hipMemcpy2D(gd_afPadScnInPad, (giScnWPad * sizeof(hipfftReal)), gd_afPadScnIn, giScnW * sizeof(hipfftReal), giScnW * sizeof(hipfftReal), giScnH, hipMemcpyDeviceToDevice));
	//Running the correlation...
	InitTim();
	//take the FFT of the scene
	InitKerTim(2);
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)gd_afPadScnInPad, (hipfftComplex *)gd_afPadScnOutPad));
	WrapKerTim("wholeFFT", 2);
	//apply kth law to scene
	InitKerTim(2);
	kthLaw << <gdBlocksWhole, gdThreadsWhole >> >(gd_afPadScnOutPad, giScnSzPad);
	WrapKerTim("wholeKth", 2);
	//initialize max PSR value
	fMaxPSR = INT_MIN;
	//First find the peak with MulCompFlts
	WrapTim("FirstPassInit");
	InitTim();
	for (iFltIndx = giNumSngCompFlt; iFltIndx < giNumOrigFlt; iFltIndx++)
	{
		for (iSzIndx = 0; iSzIndx < giNumSz; iSzIndx++)
		{
			for (iIPIndx = giBegIdxIPInFirst; iIPIndx < giEndIdxIPInFirst; iIPIndx++)
			{
				//I am not initializing gh_afArea. make sure you reach right coords.
				getWholeTplFFT(gd_afCompFlt, iIPIndx, iSzIndx, iFltIndx, gd_afPadTplIn, &gd_afPadTplOut, ghFFTplanWholeFwd, gd_afWholeTplFFT);
				//perform correlation
				Corr(gd_afPadTplOut, gdBlocksWhole, gdThreadsWhole, gd_afPadScnOutPad, giScnSzPad, gd_afMul, ghFFTplanWholeInv, gd_afCorr, gh_afArea, &iPeakIndx, &fPSR, giScnWPad, giScnHPad);
				//printf("PSR value for MulCompFlt: %f (iFltIndx = %d IPAng = %d, Sz = %d)\n", fPSR, iFltIndx, gstCompFlt.aiIPAngs[iIPIndx], gstCompFlt.aiTplCols[iSzIndx]);
				if (fPSR > fMaxPSR)
				{
					fMaxPSR = fPSR;
					iMaxIPIndx = iIPIndx;// which filter
					iMaxSzIndx = iSzIndx;
					*piMaxPeakIndx = iPeakIndx;// location of peak
				}
			}
		}
	}
#ifndef DoIPInSecond
	giBegIdxIPInSecond = iMaxIPIndx;
	giEndIdxIPInSecond = giBegIdxIPInSecond + 1;
#endif
	WrapTim("FirstPassLoop");
#ifdef CHECKRES
#ifndef DoIPInSecond
	if (bLoadScn) //if processing a scn from file(no video input), and trying IPRots in first pass
	{
		//make sure this is the last tpl
		if (iFltIndx == giNumOrigFlt && iSzIndx == giNumSz && iIPIndx == giNumIPRot)
		{
			cmpCPU(gd_afCorr, "resMulFFTInv.bin", 0, giScnSz, 0, (float)1e-6);
			cmpCPU(&fPSR, "PSR.bin", 0, 1, 1, (float)1e-6);
		}
	}
#endif
#endif
	////////SECOND PASS///////////
	InitTim();
	//we know the max IP and Sz. Now try different templates
	//copy template-size portion of the scene starting at peak point
	//	CUDA_SAFE_CALL( hipMemcpy2D( gd_afScnPartIn, giTplWMemSz, gd_afPadScnIn+iMaxPeakIndx, giScnW*sizeof(hipfftReal), giTplWMemSz, giTplH , hipMemcpyDeviceToDevice ));
	getCopyWidthHeight(*piMaxPeakIndx, piPartW, piPartH);
	int ConvertPeakIndx = *piMaxPeakIndx % 1024 + (*piMaxPeakIndx / 1024) * 640;//***
	iPartWMemSz = *piPartW * sizeof(hipfftReal);
	//make sure you initialize gd_afScnPartIn with zeros before processing each frame (if we are out of bounds, we will have a part image padded with zeros)
	InitKerTim(3);
	CUDA_SAFE_CALL(hipMemset(gd_afScnPartIn, 0, giTplMemSzReal));
	//copy the part where PSR is highest in the first pass
	CUDA_SAFE_CALL(hipMemcpy2D(gd_afScnPartIn, giTplWMemSz, gd_afPadScnIn + ConvertPeakIndx, giScnW * sizeof(hipfftReal), iPartWMemSz, *piPartH, hipMemcpyDeviceToDevice));
	WrapKerTim("MemcpyD2DPart", 3);
	//take the FFT of the scene
	InitKerTim(3);
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanPartFwd, (hipfftReal *)gd_afScnPartIn, (hipfftComplex *)gd_afScnPartOut));
	WrapKerTim("partFFT", 3);
	//apply kth law to scene
	InitKerTim(3);
	kthLaw << <gdBlocksPart, gdThreadsPart >> >(gd_afScnPartOut, giTplSz);
	WrapKerTim("partKth", 3);
	fMaxPSR = INT_MIN;
	WrapTim("SecondPassInit");
	InitTim();
	for (iFltIndx = 0; iFltIndx < giNumSngCompFlt; iFltIndx++)
	{
		for (iIPIndx = giBegIdxIPInSecond; iIPIndx < giEndIdxIPInSecond; iIPIndx++)
		{
			getPartTplFFT(gd_afCompFlt, iIPIndx, iMaxSzIndx, iFltIndx, &gd_afPadTplOut, ghFFTplanPartFwd, gd_afPartTplFFT);
			// iPeakIndx is useless here
			Corr(gd_afPadTplOut, gdBlocksPart, gdThreadsPart, gd_afScnPartOut, giTplSz, gd_afMul, ghFFTplanPartInv, gd_afCorr, gh_afArea, &iPeakIndx, &fPSR, giTplW, giTplH);
			if (fPSR > fMaxPSR)
			{
				fMaxPSR = fPSR;
				iMaxFltIndx = iFltIndx;
				iMaxIPIndx = iIPIndx;
			}
		}
	}
	WrapTim("SecondPassLoop");

#ifdef CHECKRES
#ifndef DoIPInSecond
	if (bLoadScn) //if processing a scn from file(no video input), and trying IPRots in first pass
	{
		//make sure this is the first tpl (the one before MulCompFlts)
		if (iFltIndx == giNumSngCompFlt)
		{
			cmpCPU(&fPSR, "PSRPart.bin", 0, 1, 1, (float)1e-4);
		}
	}
#endif
#endif

#ifdef KERTIM
	printf("Kernel time: %f msecs.\n", g_dTotalKerTime);
#endif
#ifdef PARTIM
	printf("GPU time: %f msecs.\n", g_dRunsOnGPUTotalTime);
	printf("\nRuntime(GPU time + Clahe): %f msecs.\n\n", g_dRunsOnGPUTotalTime + g_dClaheTime);
#endif
#ifdef ALLTIM
	CUT_SAFE_CALL(cutStopTimer(uiAllTim));
	double gpuTime = sdkGetTimerValue(uiAllTim);
	//#ifndef PARTIM
	printf("Runtime(GPU time + Clahe): %f msecs.\n", gpuTime);
	//#endif
#endif

	DisplayResults(fMaxPSR, iMaxFltIndx, iMaxIPIndx, iMaxSzIndx, file_info[0]);
	//in realis show the peak in correct position (add offset of the window in the frame)
#ifdef SHOWBOX_WHENRECOG
	if (fMaxPSR <= gfPSRTrashold) //hide the box upper left corner, if the PSR is below trashold
		*piMaxPeakIndx = 0 - ((giTplH*giScnW * 2) + giScnOffset);
#endif
	*piMaxPeakIndx = *piMaxPeakIndx + giScnOffset;
	//printf("MaxPeakIndx: %d, FrameID: %d\n", *piMaxPeakIndx, file_info[0]);
}


void ssd_fft_gpu_findBestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp)
{
	BestTpl(acScn, piMaxPeakIndx, piPartW, piPartH, file_info, ulTimeStamp);
}


void ssd_fft_gpu_returnBestTpl(unsigned char* acScn, int* piMaxPeakIndx, int* piPartW, int* piPartH, int* file_info, unsigned long ulTimeStamp, int* iSLCurFrm, int* iSLResult, char* acClipName)
{
	strcpy(gacClipName, acClipName);
	/*bool bFixDead = gbFixDead;
	CpyScnToDevAndPreProcess(acScn, gd_afPadScnIn, gbConGam, bFixDead);
	CUFFT_SAFE_CALL(hipfftExecR2C(ghFFTplanWholeFwd, (hipfftReal *)gd_afPadScnIn, (hipfftComplex *)gd_afPadScnOut));
	CUFFT_SAFE_CALL(hipfftExecC2R(ghFFTplanWholeInv, (hipfftComplex *)gd_afPadScnOut, (hipfftReal *)gd_afPadScnIn));
	float* h_acScn;
	h_acScn = (float*)malloc(giScnMemSzReal);
	CUFFT_SAFE_CALL(hipMemcpy(h_acScn, (hipfftReal*)gd_afPadScnIn, giScnMemSzReal, hipMemcpyDeviceToHost));
	for (int i = 0; i < 307200; i++)
	{
		acScn[i] = 255 * h_acScn[i];
	}
	//convertChar4ToFloatDoConGam << <gdBlocksConv, gdThreadsConv >> > (gd_ac4Scn, (float4*)d_afPadScnIn, (giScnSz / 4), bConGam);
	*/
	BestTpl(acScn, piMaxPeakIndx, piPartW, piPartH, file_info, ulTimeStamp);
	*iSLCurFrm = giSLCurFrm;
	*iSLResult = giSLResult;
}


void ssd_fft_gpu_exit() {
#ifndef US_SIGNS
#ifdef STATS
	//add the last video time	
	g_iNumVideos++;
	g_fAllVideoTime = g_fAllVideoTime + (float)(((double)g_ulLastTimeStamp - (double)g_ulFirstTimeStamp) / 1000 / 60);
	//write all video time to the file
	strcpy(g_sStatsPath, g_sStatsPathBegin);
	strcat(g_sStatsPath, "AllVideoTime.txt\0");
	g_fStatsFile = fopen(g_sStatsPath, "wb");
	if (g_fStatsFile == NULL)
		printf("Error openning stats file for measuring all video time!");
	fprintf(g_fStatsFile, "%d\t%f\t%d\t%d\t%d\t%d\t%d\n", g_iNumVideos, g_fAllVideoTime, gi16fps, gi8fps, gi5fps, gi4fps, gi0fps);
	fclose(g_fStatsFile);
#endif
#endif

	printf("Shutting down...\n");
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanWholeFwd));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanWholeInv));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanPartFwd));
	CUFFT_SAFE_CALL(hipfftDestroy(ghFFTplanPartInv));
	CUDA_SAFE_CALL(hipFree(gd_ac4Scn));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnIn));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnInPad));
	CUDA_SAFE_CALL(hipFree(gd_afScnPartIn));
	CUDA_SAFE_CALL(hipFree(gd_afScnPartOut));
	CUDA_SAFE_CALL(hipFree(gd_afCompFlt));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnOut));
	CUDA_SAFE_CALL(hipFree(gd_afPadScnOutPad));
	CUDA_SAFE_CALL(hipFree(gd_afCorr));
	CUDA_SAFE_CALL(hipFree(gd_afMul));
	CUDA_SAFE_CALL(hipFree(gd_pfMax));
	CUDA_SAFE_CALL(hipFree(gd_afBlockMaxs));
	CUDA_SAFE_CALL(hipFree(gd_piMaxIdx));
	CUDA_SAFE_CALL(hipFree(gd_aiBlockMaxIdxs));
	DestroyTplFFT(gd_afWholeTplFFT, gd_afPartTplFFT, gd_afPadTplIn, gd_afPadTplOut);
#ifdef PINNED_MEM
	hipHostFree(gh_acScn);
	hipHostFree(gh_afArea);
	hipHostFree(gstCompFlt.aiIPAngs);
	hipHostFree(gstCompFlt.aiTplCols);
	hipHostFree(gstCompFlt.aiTpl_no);
	hipHostFree(gstCompFlt.h_afData);
	hipHostFree(gastAccRes);
#else
	free(gh_acScn);
	free(gh_afArea);
	free(gstCompFlt.aiIPAngs);
	free(gstCompFlt.aiTplCols);
	free(gstCompFlt.aiTpl_no);
	free(gstCompFlt.h_afData);
	free(gastAccRes);
#endif
	//CUT_EXIT(argc, argv);
}

